#include "hip/hip_runtime.h"
//#include <cutil_inline.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <windows.h>

hipDeviceProp_t _dDeviceProp; 

// Device buffers
float* _dObjects;
float* _dSeries;
float* _dPerformances;
int*   _dFrequencies;

// Variables
int _dNbObjects;
int _dNbFrequencies;
int _dNbSeries;

/**
* @brief Initializes the testcase by 
* - Allocating the GPU memory
* - Copying the buffer from RAM to GPU memory.
* This function is executed on the Host
* @ param hBuffer Pointer to array of floats in host memory
* @ param N Number of objects in the array (an Objects consists of 6 floats)
*/
extern "C" 
void initialize_device( int NbSeries, int NbObjects, int NbFrequencies )
{
#if 0
   // Get Device properties
   int deviceCount; 
   hipGetDeviceCount(&deviceCount);

   int device(0); 
   hipGetDeviceProperties(&_dDeviceProp, device); 
#endif // 0

   // Testcase initialization
   _dNbObjects     = NbObjects;
   _dNbFrequencies = NbFrequencies;
   _dNbSeries      = NbSeries;

   // Device allocation
   hipMalloc( (void**)&_dObjects, NbSeries*NbObjects*sizeof(float) );
   hipMalloc( (void**)&_dSeries, NbSeries*sizeof(float) );
   hipMalloc( (void**)&_dFrequencies, NbFrequencies*sizeof(int) );
   hipMalloc( (void**)&_dPerformances, NbObjects*sizeof(float) );
}

/**
* @brief 
*/
__global__ void performanceStorageKernel( float* series, float* objects, int nbObjects, float* performances )
{
   // Compute the index
   unsigned int x     = blockIdx.x*blockDim.x+threadIdx.x;
   unsigned int y     = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int index = (y*blockDim.x) + x;

   int objectsIndex = index*nbObjects;

   // Compute performance
   __shared__ float localPerformance[2];
   localPerformance[0] = 1.0; // current performance
   localPerformance[1] = 0.0; // previous performance
   for( int i(0); i<nbObjects; ++i ) {
      localPerformance[1] = localPerformance[0];
      localPerformance[0] = (1.0+objects[objectsIndex+i])*localPerformance[1];

      if( index == 0 ) performances[i] = localPerformance[0];
   }
   // Store performance
   series[index] = localPerformance[0] - 1.0;
}

/**
* @brief Kernel function to be executed on the GPU
* @param ptr Pointer to an array of floats stored in GPU memory
*/
__global__ void frequenciesKernel( float* series, float range, int* frequencies, int nbFrequencies  )
{
   // Compute the index
   unsigned int x     = blockIdx.x*blockDim.x+threadIdx.x;
   unsigned int y     = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int index = (y*blockDim.x) + x;

   float v = series[index]-(-range/2);
   int position = (v/(range/nbFrequencies));
   atomicAdd(&frequencies[position],1);
}

/**
* @brief Run the kernel on the GPU. 
* This function is executed on the Host.
*/
extern "C" 
void run_kernel( float range, float mu, float sigma, int random, int nbThreadsPerBlock )
{
#if 0
   // Get Device properties
   int deviceCount; 
   hipGetDeviceCount(&deviceCount);
   int device(0); 
   hipGetDeviceProperties(&_dDeviceProp, device); 
#endif // 0
   long t = 0;
   t = GetTickCount();

   // Create pseudo-random number generator
   hiprandGenerator_t gen;
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
   hiprandSetPseudoRandomGeneratorSeed(gen, random);
   hiprandGenerateNormal(gen, _dObjects, _dNbSeries*_dNbObjects, mu, sigma );
   hiprandDestroyGenerator(gen);

   dim3 block(8, 8, 1);
   int gridDim = (int)sqrt(float(_dNbSeries));
   dim3 gridPerformances( gridDim/block.x, gridDim/block.y, 1);
   performanceStorageKernel<<<gridPerformances,block>>>(_dSeries, _dObjects, _dNbObjects, _dPerformances );

   // Reset memory
   hipMemset( _dFrequencies, 0, _dNbFrequencies*sizeof(int) );

   // compute Frequencies 
   gridDim = (int)sqrt(float(_dNbSeries));
   dim3 gridFrequencies( gridDim/block.x, gridDim/block.y, 1);
   frequenciesKernel<<<gridFrequencies,block>>>( _dSeries, range, _dFrequencies, _dNbFrequencies );

   t = GetTickCount() - t;
   t = 1;
}

/*
* This function is executed on the Host.
* @brief Copy the data back to the Host and releases the buffer on
* GPU device.
*/ 
extern "C" 
void host_to_device( float* hOutObjects )
{
}

/*
* This function is executed on the Host.
* @brief Copy the data back to the Host and releases the buffer on
* GPU device.
* This function is executed on the Host
* @ param hBuffer Pointer to array of floats in host memory
* @ param N Number of objects in the array (an Objects consists of 6 floats)
*/
extern "C" 
void device_to_host( int* hOutFrequencies, float* hOutPerformances, float* hOutSeries )
{
   hipMemcpy( hOutFrequencies,  _dFrequencies,  _dNbFrequencies*sizeof(int), hipMemcpyDeviceToHost);
   hipMemcpy( hOutPerformances, _dPerformances, _dNbObjects*sizeof(float),   hipMemcpyDeviceToHost);
}

/*
* This function is executed on the Host.
* @brief Releases the buffer on GPU device.
* This function is executed on the Host
*/
extern "C" 
void destroy_device()
{
   hipFree( _dObjects );
   hipFree( _dFrequencies );
   hipFree( _dSeries );
   hipFree( _dPerformances );
}
